#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include "open3d/core/Dispatch.h"
#include "open3d/core/Dtype.h"
#include "open3d/core/MemoryManager.h"
#include "open3d/core/SizeVector.h"
#include "open3d/core/Tensor.h"
#include "open3d/core/hashmap/Hashmap.h"
#include "open3d/core/kernel/CUDALauncher.cuh"
#include "open3d/core/kernel/GeneralEW.h"
#include "open3d/core/kernel/GeneralEWMacros.h"
#include "open3d/core/kernel/GeneralIndexer.h"
#include "open3d/utility/Console.h"

namespace open3d {
namespace core {
namespace kernel {

void CUDAUnprojectKernel(const std::unordered_map<std::string, Tensor>& srcs,
                         std::unordered_map<std::string, Tensor>& dsts) {
    static std::vector<std::string> src_attrs = {
            "depth", "intrinsics", "depth_scale", "depth_max", "stride"};
    for (auto& k : src_attrs) {
        if (srcs.count(k) == 0) {
            utility::LogError(
                    "[CUDAUnprojectKernel] expected Tensor {} in srcs, but "
                    "did not receive",
                    k);
        }
    }

    // Input
    Tensor depth = srcs.at("depth").To(core::Dtype::Float32);
    Tensor intrinsics = srcs.at("intrinsics").To(core::Dtype::Float32);
    float depth_scale = srcs.at("depth_scale").Item<float>();
    float depth_max = srcs.at("depth_max").Item<float>();
    int64_t stride = srcs.at("stride").Item<int64_t>();

    NDArrayIndexer depth_ndi(depth, 2);
    TransformIndexer ti(intrinsics);

    // Output
    int64_t rows_strided = depth_ndi.GetShape(0) / stride;
    int64_t cols_strided = depth_ndi.GetShape(1) / stride;
    Tensor points({rows_strided * cols_strided, 3}, core::Dtype::Float32,
                  depth.GetDevice());
    Tensor count(std::vector<int>{0}, {}, core::Dtype::Int32,
                 depth.GetDevice());
    float* points_ptr = static_cast<float*>(points.GetDataPtr());
    int* count_ptr = static_cast<int*>(count.GetDataPtr());

    // Workload
    int64_t n = rows_strided * cols_strided;

    CUDALauncher::LaunchGeneralKernel(
            n, [=] OPEN3D_DEVICE(int64_t workload_idx) {
                int64_t y = (workload_idx / cols_strided) * stride;
                int64_t x = (workload_idx % cols_strided) * stride;

                int64_t workload_depth;
                depth_ndi.CoordToWorkload(x, y, &workload_depth);
                float d = *static_cast<float*>(depth_ndi.GetDataPtrFromWorkload(
                                  workload_depth)) /
                          depth_scale;
                if (d > 0 && d < depth_max) {
                    int idx = atomicAdd(count_ptr, 1);
                    float* vertex = points_ptr + 3 * idx;
                    ti.Unproject(static_cast<float>(x), static_cast<float>(y),
                                 d, vertex + 0, vertex + 1, vertex + 2);
                }
            });

    int total_pts_count = count.Item<int>();
    dsts.emplace("points", points.Slice(0, 0, total_pts_count));
}

/// Dummy kernel launch: global hashmap calls
void CUDATSDFTouchKernel(const std::unordered_map<std::string, Tensor>& srcs,
                         std::unordered_map<std::string, Tensor>& dsts) {
    static std::vector<std::string> src_attrs = {
            "points",
            "voxel_size",
            "resolution",
    };

    for (auto& k : src_attrs) {
        if (srcs.count(k) == 0) {
            utility::LogError(
                    "[CUDATSDFTouchKernel] expected Tensor {} in srcs, but "
                    "did not receive",
                    k);
        }
    }

    Tensor pcd = srcs.at("points");
    float voxel_size = srcs.at("voxel_size").Item<float>();
    int64_t resolution = srcs.at("resolution").Item<int64_t>();
    float block_size = voxel_size * resolution;

    float sdf_trunc = srcs.at("sdf_trunc").Item<float>();

    Device device = pcd.GetDevice();

    int64_t n = pcd.GetShape()[0];
    float* pcd_ptr = static_cast<float*>(pcd.GetDataPtr());

    Tensor block_coordi({8 * n, 3}, Dtype::Int32, device);
    int* block_coordi_ptr = static_cast<int*>(block_coordi.GetDataPtr());
    Tensor count(std::vector<int>{0}, {}, Dtype::Int32, device);
    int* count_ptr = static_cast<int*>(count.GetDataPtr());

    CUDALauncher::LaunchGeneralKernel(
            n, [=] OPEN3D_DEVICE(int64_t workload_idx) {
                float x = pcd_ptr[3 * workload_idx + 0];
                float y = pcd_ptr[3 * workload_idx + 1];
                float z = pcd_ptr[3 * workload_idx + 2];

                int xb_lo = static_cast<int>((x - sdf_trunc) / block_size);
                int xb_hi = static_cast<int>((x + sdf_trunc) / block_size);
                int yb_lo = static_cast<int>((y - sdf_trunc) / block_size);
                int yb_hi = static_cast<int>((y + sdf_trunc) / block_size);
                int zb_lo = static_cast<int>((z - sdf_trunc) / block_size);
                int zb_hi = static_cast<int>((z + sdf_trunc) / block_size);
                for (int64_t xb = xb_lo; xb <= xb_hi; ++xb) {
                    for (int64_t yb = yb_lo; yb <= yb_hi; ++yb) {
                        for (int64_t zb = zb_lo; zb <= zb_hi; ++zb) {
                            int idx = atomicAdd(count_ptr, 1);
                            block_coordi_ptr[3 * idx + 0] = xb;
                            block_coordi_ptr[3 * idx + 1] = yb;
                            block_coordi_ptr[3 * idx + 2] = zb;
                        }
                    }
                }
            });

    int total_block_count = count.Item<int>();
    block_coordi = block_coordi.Slice(0, 0, total_block_count);
    core::Hashmap pcd_block_hashmap(
            total_block_count,
            core::Dtype(core::Dtype::DtypeCode::Object,
                        core::Dtype::Int32.ByteSize() * 3, "_hash_k"),
            core::Dtype::Int32, device);
    core::Tensor block_addrs, block_masks;
    pcd_block_hashmap.Activate(block_coordi.Slice(0, 0, count.Item<int>()),
                               block_addrs, block_masks);
    dsts.emplace("block_coords", block_coordi.IndexGet({block_masks}));
}

void CUDATSDFIntegrateKernel(
        const std::unordered_map<std::string, Tensor>& srcs,
        std::unordered_map<std::string, Tensor>& dsts) {
    // Decode input tensors
    static std::vector<std::string> src_attrs = {
            "depth",      "indices",    "block_keys",
            "intrinsics", "extrinsics", "resolution",
            "voxel_size", "sdf_trunc",  "depth_scale",
    };
    for (auto& k : src_attrs) {
        if (srcs.count(k) == 0) {
            utility::LogError(
                    "[CUDATSDFIntegrateKernel] expected Tensor {} in srcs, but "
                    "did not receive",
                    k);
        }
    }

    Tensor depth = srcs.at("depth").To(core::Dtype::Float32);
    Tensor indices = srcs.at("indices");
    Tensor block_keys = srcs.at("block_keys");
    Tensor block_values = dsts.at("block_values");

    // Transforms
    Tensor intrinsics = srcs.at("intrinsics").To(core::Dtype::Float32);
    Tensor extrinsics = srcs.at("extrinsics").To(core::Dtype::Float32);

    // Parameters
    int64_t resolution = srcs.at("resolution").Item<int64_t>();
    int64_t resolution3 = resolution * resolution * resolution;

    float voxel_size = srcs.at("voxel_size").Item<float>();
    float sdf_trunc = srcs.at("sdf_trunc").Item<float>();
    float depth_scale = srcs.at("depth_scale").Item<float>();

    // Shape / transform indexers, no data involved
    NDArrayIndexer voxel_indexer({resolution, resolution, resolution});
    TransformIndexer transform_indexer(intrinsics, extrinsics, voxel_size);

    // Real data indexer
    NDArrayIndexer image_indexer(depth, 2);
    NDArrayIndexer voxel_block_buffer_indexer(block_values, 4);

    // Plain arrays that does not require indexers
    int64_t* indices_ptr = static_cast<int64_t*>(indices.GetDataPtr());
    int* block_keys_ptr = static_cast<int*>(block_keys.GetDataPtr());

    int64_t n = indices.GetShape()[0] * resolution3;
    CUDALauncher::LaunchGeneralKernel(n, [=] OPEN3D_HOST_DEVICE(
                                                 int64_t workload_idx) {
        // Natural index (0, N) -> (block_idx, voxel_idx)
        int64_t block_idx = indices_ptr[workload_idx / resolution3];
        int64_t voxel_idx = workload_idx % resolution3;

        /// Coordinate transform
        // block_idx -> (x_block, y_block, z_block)
        int64_t xb = static_cast<int64_t>(block_keys_ptr[block_idx * 3 + 0]);
        int64_t yb = static_cast<int64_t>(block_keys_ptr[block_idx * 3 + 1]);
        int64_t zb = static_cast<int64_t>(block_keys_ptr[block_idx * 3 + 2]);

        // voxel_idx -> (x_voxel, y_voxel, z_voxel)
        int64_t xv, yv, zv;
        voxel_indexer.WorkloadToCoord(voxel_idx, &xv, &yv, &zv);

        // coordinate in world (in voxel)
        int64_t x = (xb * resolution + xv);
        int64_t y = (yb * resolution + yv);
        int64_t z = (zb * resolution + zv);

        // coordinate in camera (in voxel -> in meter)
        float xc, yc, zc, u, v;
        transform_indexer.RigidTransform(static_cast<float>(x),
                                         static_cast<float>(y),
                                         static_cast<float>(z), &xc, &yc, &zc);

        // coordinate in image (in pixel)
        transform_indexer.Project(xc, yc, zc, &u, &v);
        if (!image_indexer.InBoundary(u, v)) {
            return;
        }

        /// Associate image workload and compute SDF
        int64_t workload_image;
        image_indexer.CoordToWorkload(static_cast<int64_t>(u),
                                      static_cast<int64_t>(v), &workload_image);
        float depth =
                *static_cast<const float*>(
                        image_indexer.GetDataPtrFromWorkload(workload_image)) /
                depth_scale;
        float sdf = depth - zc;
        if (depth <= 0 || zc <= 0 || sdf < -sdf_trunc) {
            return;
        }
        sdf = sdf < sdf_trunc ? sdf : sdf_trunc;
        sdf /= sdf_trunc;

        /// Associate voxel workload and update TSDF/Weights
        int64_t workload_voxel;
        voxel_block_buffer_indexer.CoordToWorkload(xv, yv, zv, block_idx,
                                                   &workload_voxel);
        float* voxel_ptr = static_cast<float*>(
                voxel_block_buffer_indexer.GetDataPtrFromWorkload(
                        workload_voxel));

        float tsdf_sum = voxel_ptr[0];
        float weight_sum = voxel_ptr[1];
        voxel_ptr[0] = (weight_sum * tsdf_sum + sdf) / (weight_sum + 1);
        voxel_ptr[1] = weight_sum + 1;
    });
}

void CUDASurfaceExtractionKernel(
        const std::unordered_map<std::string, Tensor>& srcs,
        std::unordered_map<std::string, Tensor>& dsts) {
    // Decode input tensors
    static std::vector<std::string> src_attrs = {
            "indices",      "nb_indices", "nb_masks",   "block_keys",
            "block_values", "voxel_size", "resolution",
    };
    for (auto& k : src_attrs) {
        if (srcs.count(k) == 0) {
            utility::LogError(
                    "[CPUTSDFIntegrateKernel] expected Tensor {} in srcs, but "
                    "did not receive",
                    k);
        }
    }
    utility::LogInfo("surface extraction starts");

    Tensor indices = srcs.at("indices");
    Tensor nb_indices = srcs.at("nb_indices");
    Tensor nb_masks = srcs.at("nb_masks");
    Tensor block_keys = srcs.at("block_keys");
    Tensor block_values = srcs.at("block_values");

    // Parameters
    int64_t resolution = srcs.at("resolution").Item<int64_t>();
    int64_t resolution3 = resolution * resolution * resolution;

    float voxel_size = srcs.at("voxel_size").Item<float>();

    // Shape / transform indexers, no data involved
    NDArrayIndexer voxel_indexer({resolution, resolution, resolution});

    // Real data indexer
    NDArrayIndexer voxel_block_buffer_indexer(block_values, 4);

    // Plain arrays that does not require indexers
    int64_t* nb_indices_ptr = static_cast<int64_t*>(nb_indices.GetDataPtr());
    bool* nb_masks_ptr = static_cast<bool*>(nb_masks.GetDataPtr());
    int64_t* indices_ptr = static_cast<int64_t*>(indices.GetDataPtr());
    int* block_keys_ptr = static_cast<int*>(block_keys.GetDataPtr());

    int n_blocks = indices.GetShape()[0];
    int64_t n = n_blocks * resolution3;

    // Output
    core::Tensor count(std::vector<int>{0}, {}, core::Dtype::Int32,
                       block_values.GetDevice());
    core::Tensor points({std::min(n * 3, int64_t(10000000)), 3},
                        core::Dtype::Float32, block_values.GetDevice());
    int* count_ptr = static_cast<int*>(count.GetDataPtr());
    float* points_ptr = static_cast<float*>(points.GetDataPtr());

    CUDALauncher::LaunchGeneralKernel(n, [=] OPEN3D_DEVICE(
                                                 int64_t workload_idx) {
        // Natural index (0, N) -> (block_idx, voxel_idx)
        int64_t workload_block_idx = workload_idx / resolution3;
        int64_t block_idx = indices_ptr[workload_block_idx];
        int64_t voxel_idx = workload_idx % resolution3;

        /// Coordinate transform
        // block_idx -> (x_block, y_block, z_block)
        int64_t xb = static_cast<int64_t>(block_keys_ptr[block_idx * 3 + 0]);
        int64_t yb = static_cast<int64_t>(block_keys_ptr[block_idx * 3 + 1]);
        int64_t zb = static_cast<int64_t>(block_keys_ptr[block_idx * 3 + 2]);

        // voxel_idx -> (x_voxel, y_voxel, z_voxel)
        int64_t xv, yv, zv;
        voxel_indexer.WorkloadToCoord(voxel_idx, &xv, &yv, &zv);
        int64_t workload_voxel;
        voxel_block_buffer_indexer.CoordToWorkload(xv, yv, zv, block_idx,
                                                   &workload_voxel);
        float* voxel_ptr = static_cast<float*>(
                voxel_block_buffer_indexer.GetDataPtrFromWorkload(
                        workload_voxel));
        float tsdf_o = voxel_ptr[0];
        float weight_o = voxel_ptr[1];
        if (weight_o == 0) return;

        int64_t x = xb * resolution + xv;
        int64_t y = yb * resolution + yv;
        int64_t z = zb * resolution + zv;

        for (int i = 0; i < 3; ++i) {
            int64_t xv_i = xv + int64_t(i == 0);
            int64_t yv_i = yv + int64_t(i == 1);
            int64_t zv_i = zv + int64_t(i == 2);

            int64_t dxb = xv_i / resolution;
            int64_t dyb = yv_i / resolution;
            int64_t dzb = zv_i / resolution;

            int64_t nb_idx = (dxb + 1) + (dyb + 1) * 3 + (dzb + 1) * 9;

            if (nb_indices_ptr[13 * n_blocks + workload_block_idx] !=
                block_idx) {
                printf("wrong!\n");
            }
            bool block_mask_i =
                    nb_masks_ptr[nb_idx * n_blocks + workload_block_idx];
            if (!block_mask_i) continue;

            int64_t block_idx_i =
                    nb_indices_ptr[nb_idx * n_blocks + workload_block_idx];
            int64_t workload_voxel_i;
            voxel_block_buffer_indexer.CoordToWorkload(
                    xv_i - dxb * resolution, yv_i - dyb * resolution,
                    zv_i - dzb * resolution, block_idx_i, &workload_voxel_i);
            // printf("%ld %ld %ld at %d: %ld %ld %ld\n", xv_i, yv_i, zv_i, i,
            //        xv_i - dxb * resolution, yv_i - dyb * resolution,
            //        zv_i - dzb * resolution);
            float* voxel_ptr_i = static_cast<float*>(
                    voxel_block_buffer_indexer.GetDataPtrFromWorkload(
                            workload_voxel_i));

            float tsdf_i = voxel_ptr_i[0];
            float weight_i = voxel_ptr_i[1];

            if (weight_i > 0 && tsdf_i * tsdf_o < 0) {
                float ratio = (0 - tsdf_o) / (tsdf_i - tsdf_o);

                int idx = atomicAdd(count_ptr, 1);

                points_ptr[idx * 3 + 0] =
                        voxel_size * (x + ratio * int(i == 0));
                points_ptr[idx * 3 + 1] =
                        voxel_size * (y + ratio * int(i == 1));
                points_ptr[idx * 3 + 2] =
                        voxel_size * (z + ratio * int(i == 2));
            }
        }
    });

    int total_count = count.Item<int>();
    dsts.emplace("points", points.Slice(0, 0, total_count));
    utility::LogInfo("surface extraction finished");
}

void CUDAMarchingCubesKernel(
        const std::unordered_map<std::string, Tensor>& srcs,
        std::unordered_map<std::string, Tensor>& dsts) {
    // Decode input tensors
    static std::vector<std::string> src_attrs = {
            "indices",    "inv_indices",  "nb_indices", "nb_masks",
            "block_keys", "block_values", "voxel_size", "resolution",
    };
    for (auto& k : src_attrs) {
        if (srcs.count(k) == 0) {
            utility::LogError(
                    "[CUDAMarchingCubesKernel] expected Tensor {} in srcs, but "
                    "did not receive",
                    k);
        }
    }
    utility::LogInfo("surface extraction starts");

    CUDACachedMemoryManager::ReleaseCache();

    Tensor indices = srcs.at("indices");
    Tensor inv_indices = srcs.at("inv_indices");
    Tensor nb_indices = srcs.at("nb_indices");
    Tensor nb_masks = srcs.at("nb_masks");
    Tensor block_keys = srcs.at("block_keys");
    Tensor block_values = srcs.at("block_values");

    // Parameters
    int64_t resolution = srcs.at("resolution").Item<int64_t>();
    int64_t resolution3 = resolution * resolution * resolution;

    float voxel_size = srcs.at("voxel_size").Item<float>();

    // Shape / transform indexers, no data involved
    NDArrayIndexer voxel_indexer({resolution, resolution, resolution});

    // Output
    int n_blocks = indices.GetShape()[0];
    core::Tensor mesh_structure = core::Tensor::Zeros(
            {n_blocks, resolution, resolution, resolution, 4},
            core::Dtype::Int32, block_keys.GetDevice());

    // Real data indexer
    NDArrayIndexer voxel_block_buffer_indexer(block_values, 4);
    NDArrayIndexer mesh_structure_indexer(mesh_structure, 4);

    // Plain arrays that does not require indexers
    int64_t* nb_indices_ptr = static_cast<int64_t*>(nb_indices.GetDataPtr());
    bool* nb_masks_ptr = static_cast<bool*>(nb_masks.GetDataPtr());
    int64_t* indices_ptr = static_cast<int64_t*>(indices.GetDataPtr());
    int64_t* inv_indices_ptr = static_cast<int64_t*>(inv_indices.GetDataPtr());
    int* block_keys_ptr = static_cast<int*>(block_keys.GetDataPtr());

    int64_t n = n_blocks * resolution3;

    // Pass 0: analyze mesh structure, set up one-on-one correspondences
    CUDALauncher::LaunchGeneralKernel(n, [=] OPEN3D_DEVICE(
                                                 int64_t workload_idx) {
        // Natural index (0, N) -> (block_idx, voxel_idx)
        int64_t workload_block_idx = workload_idx / resolution3;
        int64_t voxel_idx = workload_idx % resolution3;

        // voxel_idx -> (x_voxel, y_voxel, z_voxel)
        int64_t xv, yv, zv;
        voxel_indexer.WorkloadToCoord(voxel_idx, &xv, &yv, &zv);

        // Check per-vertex sign in the cube to determine cube type
        int table_idx = 0;
        for (int i = 0; i < 8; ++i) {
            int64_t xv_i = xv + vtx_shifts[i][0];
            int64_t yv_i = yv + vtx_shifts[i][1];
            int64_t zv_i = zv + vtx_shifts[i][2];

            int64_t dxb = xv_i / resolution;
            int64_t dyb = yv_i / resolution;
            int64_t dzb = zv_i / resolution;

            int64_t nb_idx = (dxb + 1) + (dyb + 1) * 3 + (dzb + 1) * 9;

            bool block_mask_i =
                    nb_masks_ptr[nb_idx * n_blocks + workload_block_idx];
            if (!block_mask_i) return;

            int64_t block_idx_i =
                    nb_indices_ptr[nb_idx * n_blocks + workload_block_idx];
            int64_t workload_voxel_i;
            voxel_block_buffer_indexer.CoordToWorkload(
                    xv_i - dxb * resolution, yv_i - dyb * resolution,
                    zv_i - dzb * resolution, block_idx_i, &workload_voxel_i);
            float* voxel_ptr_i = static_cast<float*>(
                    voxel_block_buffer_indexer.GetDataPtrFromWorkload(
                            workload_voxel_i));

            float tsdf_i = voxel_ptr_i[0];
            float weight_i = voxel_ptr_i[1];
            if (weight_i == 0) return;

            table_idx |= ((tsdf_i < 0) ? (1 << i) : 0);
        }

        int64_t workload_mesh_struct_idx;
        mesh_structure_indexer.CoordToWorkload(xv, yv, zv, workload_block_idx,
                                               &workload_mesh_struct_idx);
        int* mesh_struct_ptr =
                static_cast<int*>(mesh_structure_indexer.GetDataPtrFromWorkload(
                        workload_mesh_struct_idx));
        mesh_struct_ptr[3] = table_idx;

        if (table_idx == 0 || table_idx == 255) return;

        // Check per-edge sign in the cube to determine cube type
        int edges_with_vertices = edge_table[table_idx];
        for (int i = 0; i < 12; ++i) {
            if (edges_with_vertices & (1 << i)) {
                int64_t xv_i = xv + edge_shifts[i][0];
                int64_t yv_i = yv + edge_shifts[i][1];
                int64_t zv_i = zv + edge_shifts[i][2];
                int edge_i = edge_shifts[i][3];

                int dxb = xv_i / resolution;
                int dyb = yv_i / resolution;
                int dzb = zv_i / resolution;

                int nb_idx = (dxb + 1) + (dyb + 1) * 3 + (dzb + 1) * 9;

                int64_t block_idx_i =
                        nb_indices_ptr[nb_idx * n_blocks + workload_block_idx];
                int64_t workload_mesh_struct_i;
                mesh_structure_indexer.CoordToWorkload(
                        xv_i - dxb * resolution, yv_i - dyb * resolution,
                        zv_i - dzb * resolution, inv_indices_ptr[block_idx_i],
                        &workload_mesh_struct_i);
                if (indices_ptr[inv_indices_ptr[block_idx_i]] != block_idx_i) {
                    printf("inv indices error!\n");
                }
                int* mesh_struct_ptr_i = static_cast<int*>(
                        mesh_structure_indexer.GetDataPtrFromWorkload(
                                workload_mesh_struct_i));

                // Non-atomic write, but we are safe
                mesh_struct_ptr_i[edge_i] = -1;
            }
        }
    });

    // Pass 1: allocate and assign vertices with normals
    core::Tensor vtx_count(std::vector<int>{0}, {}, core::Dtype::Int32,
                           block_values.GetDevice());
    core::Tensor vertices({std::min(n * 3, int64_t(5000000)), 3},
                          core::Dtype::Float32, block_values.GetDevice());
    core::Tensor normals({std::min(n * 3, int64_t(5000000)), 3},
                         core::Dtype::Float32, block_values.GetDevice());
    int* vtx_count_ptr = static_cast<int*>(vtx_count.GetDataPtr());
    float* vertices_ptr = static_cast<float*>(vertices.GetDataPtr());
    float* normals_ptr = static_cast<float*>(normals.GetDataPtr());
    CUDALauncher::LaunchGeneralKernel(n, [=] OPEN3D_DEVICE(
                                                 int64_t workload_idx) {
        // Natural index (0, N) -> (block_idx, voxel_idx)
        int64_t workload_block_idx = workload_idx / resolution3;
        int64_t block_idx = indices_ptr[workload_block_idx];
        int64_t voxel_idx = workload_idx % resolution3;

        // block_idx -> (x_block, y_block, z_block)
        int64_t xb = static_cast<int64_t>(block_keys_ptr[block_idx * 3 + 0]);
        int64_t yb = static_cast<int64_t>(block_keys_ptr[block_idx * 3 + 1]);
        int64_t zb = static_cast<int64_t>(block_keys_ptr[block_idx * 3 + 2]);

        // voxel_idx -> (x_voxel, y_voxel, z_voxel)
        int64_t xv, yv, zv;
        voxel_indexer.WorkloadToCoord(voxel_idx, &xv, &yv, &zv);

        // global coordinate (in voxels)
        int64_t x = xb * resolution + xv;
        int64_t y = yb * resolution + yv;
        int64_t z = zb * resolution + zv;

        // Obtain voxel's mesh struct ptr
        int64_t workload_mesh_struct_idx;
        mesh_structure_indexer.CoordToWorkload(xv, yv, zv, workload_block_idx,
                                               &workload_mesh_struct_idx);
        int* mesh_struct_ptr =
                static_cast<int*>(mesh_structure_indexer.GetDataPtrFromWorkload(
                        workload_mesh_struct_idx));

        // Early quit -- no allocated vertex to compute
        if (mesh_struct_ptr[0] != -1 && mesh_struct_ptr[1] != -1 &&
            mesh_struct_ptr[2] != -1) {
            return;
        }

        // Obtain voxel ptr
        int64_t workload_voxel_idx;
        voxel_block_buffer_indexer.CoordToWorkload(xv, yv, zv, block_idx,
                                                   &workload_voxel_idx);
        float* voxel_ptr = static_cast<float*>(
                voxel_block_buffer_indexer.GetDataPtrFromWorkload(
                        workload_voxel_idx));
        float tsdf_o = voxel_ptr[0];
        if (voxel_ptr[1] == 0) {
            printf("voxel weight error!\n");
        }

        // Normal buffers
        float n_o[3], n_e[3];

        // Offset vertex coordinates (plus / minus one voxel)
        int64_t xvs[2], yvs[2], zvs[2];
        // Delta block coordinates (unchanged or plus / minus one block)
        int64_t dxbs[2], dybs[2], dzbs[2];
        // TSDF
        float tsdfs[2];

        // First compute normal at origin
        for (int axis = 0; axis < 3; ++axis) {
            xvs[1] = xv + int(axis == 0);
            yvs[1] = yv + int(axis == 1);
            zvs[1] = zv + int(axis == 2);

            xvs[0] = xv - int(axis == 0);
            yvs[0] = yv - int(axis == 1);
            zvs[0] = zv - int(axis == 2);

            dxbs[1] = xvs[1] / resolution;
            dybs[1] = xvs[1] / resolution;
            dzbs[1] = zvs[1] / resolution;

            dxbs[0] = xvs[0] >= 0 ? 0 : -1;
            dybs[0] = yvs[0] >= 0 ? 0 : -1;
            dzbs[0] = zvs[0] >= 0 ? 0 : -1;

            for (int k = 0; k < 2; ++k) {
                int64_t nb_idx_k =
                        (dxbs[k] + 1) + (dybs[k] + 1) * 3 + (dzbs[k] + 1) * 9;
                bool block_mask_k =
                        nb_masks_ptr[nb_idx_k * n_blocks + workload_block_idx];
                int64_t block_idx_k = nb_indices_ptr[nb_idx_k * n_blocks +
                                                     workload_block_idx];
                int64_t workload_voxel_k;
                voxel_block_buffer_indexer.CoordToWorkload(
                        xvs[k] - dxbs[k] * resolution,
                        yvs[k] - dybs[k] * resolution,
                        zvs[k] - dzbs[k] * resolution, block_idx_k,
                        &workload_voxel_k);
                float* voxel_ptr_k = static_cast<float*>(
                        voxel_block_buffer_indexer.GetDataPtrFromWorkload(
                                workload_voxel_k));
                tsdfs[k] = block_mask_k ? voxel_ptr_k[0] : 0;
            }
            n_o[axis] = (tsdfs[1] - tsdfs[0]) / (2 * voxel_size);
        }

        // Enumerate 3 edges in the voxel
        for (int e = 0; e < 3; ++e) {
            int vertex_idx = mesh_struct_ptr[e];
            if (vertex_idx != -1) continue;

            int64_t xv_e = xv + int(e == 0);
            int64_t yv_e = yv + int(e == 1);
            int64_t zv_e = zv + int(e == 2);

            int dxb = xv_e / resolution;
            int dyb = yv_e / resolution;
            int dzb = zv_e / resolution;

            // First query tsdf
            int64_t nb_idx = (dxb + 1) + (dyb + 1) * 3 + (dzb + 1) * 9;

            bool block_mask_e =
                    nb_masks_ptr[nb_idx * n_blocks + workload_block_idx];
            if (!block_mask_e) {
                printf("edge: block mask error!\n");
            }

            int64_t block_idx_e =
                    nb_indices_ptr[nb_idx * n_blocks + workload_block_idx];
            int64_t workload_voxel_e;
            voxel_block_buffer_indexer.CoordToWorkload(
                    xv_e - dxb * resolution, yv_e - dyb * resolution,
                    zv_e - dzb * resolution, block_idx_e, &workload_voxel_e);
            float* voxel_ptr_e = static_cast<float*>(
                    voxel_block_buffer_indexer.GetDataPtrFromWorkload(
                            workload_voxel_e));
            float tsdf_e = voxel_ptr_e[0];
            if (voxel_ptr_e[1] == 0) {
                printf("edge: weight error!\n");
            }
            if (tsdf_e * tsdf_o > 0) {
                printf("tsdf error: %f %f\n", tsdf_e, tsdf_o);
                return;
            }

            // Then compute normals
            for (int axis = 0; axis < 3; ++axis) {
                xvs[1] = xv_e + int(axis == 0);
                yvs[1] = yv_e + int(axis == 1);
                zvs[1] = zv_e + int(axis == 2);

                xvs[0] = xv_e - int(axis == 0);
                yvs[0] = yv_e - int(axis == 1);
                zvs[0] = zv_e - int(axis == 2);

                dxbs[1] = xvs[1] / resolution;
                dybs[1] = xvs[1] / resolution;
                dzbs[1] = zvs[1] / resolution;

                dxbs[0] = xvs[0] >= 0 ? 0 : -1;
                dybs[0] = yvs[0] >= 0 ? 0 : -1;
                dzbs[0] = zvs[0] >= 0 ? 0 : -1;

                for (int k = 0; k < 2; ++k) {
                    int64_t nb_idx_k = (dxbs[k] + 1) + (dybs[k] + 1) * 3 +
                                       (dzbs[k] + 1) * 9;
                    bool block_mask_k = nb_masks_ptr[nb_idx_k * n_blocks +
                                                     workload_block_idx];
                    int64_t block_idx_k = nb_indices_ptr[nb_idx_k * n_blocks +
                                                         workload_block_idx];
                    int64_t workload_voxel_k;
                    voxel_block_buffer_indexer.CoordToWorkload(
                            xvs[k] - dxbs[k] * resolution,
                            yvs[k] - dybs[k] * resolution,
                            zvs[k] - dzbs[k] * resolution, block_idx_k,
                            &workload_voxel_k);
                    float* voxel_ptr_k = static_cast<float*>(
                            voxel_block_buffer_indexer.GetDataPtrFromWorkload(
                                    workload_voxel_k));
                    tsdfs[k] = block_mask_k ? voxel_ptr_k[0] : 0;
                }
                n_e[axis] = (tsdfs[1] - tsdfs[0]) / (2 * voxel_size);
            }

            float ratio = (0 - tsdf_o) / (tsdf_e - tsdf_o);

            int idx = atomicAdd(vtx_count_ptr, 1);
            mesh_struct_ptr[e] = idx;
            /// printf("%d\n", idx);

            float ratio_x = ratio * int(e == 0);
            float ratio_y = ratio * int(e == 1);
            float ratio_z = ratio * int(e == 2);

            vertices_ptr[3 * idx + 0] = voxel_size * (x + ratio_x);
            vertices_ptr[3 * idx + 1] = voxel_size * (y + ratio_y);
            vertices_ptr[3 * idx + 2] = voxel_size * (z + ratio_z);

            float nx = n_o[0] +
                       0.00001 * n_e[0];  // * (1 - ratio) + n_e[0] * (ratio);
            float ny = n_o[1];            // * (1 - ratio) + n_e[1] * (ratio);
            float nz = n_o[2];            // * (1 - ratio) + n_e[2] * (ratio);
            float norm = sqrtf(nx * nx + ny * ny + nz * nz);

            normals_ptr[3 * idx + 0] = nx / norm;
            normals_ptr[3 * idx + 1] = ny / norm;
            normals_ptr[3 * idx + 2] = nz / norm;
        }
    });

    int total_vtx_count = vtx_count.Item<int>();
    utility::LogInfo("Total vertex count = {}", total_vtx_count);
    vertices = vertices.Slice(0, 0, total_vtx_count);
    normals = normals.Slice(0, 0, total_vtx_count);
    dsts.emplace("vertices", vertices);
    dsts.emplace("normals", normals);

    // Pass 2: connect vertices
    core::Tensor triangle_count(std::vector<int>{0}, {}, core::Dtype::Int32,
                                block_values.GetDevice());
    core::Tensor triangles({std::min(total_vtx_count * 3, 8000000), 3},
                           core::Dtype::Int64, block_values.GetDevice());
    int* tri_count_ptr = static_cast<int*>(triangle_count.GetDataPtr());
    int64_t* triangles_ptr = static_cast<int64_t*>(triangles.GetDataPtr());

    CUDALauncher::LaunchGeneralKernel(n, [=] OPEN3D_DEVICE(
                                                 int64_t workload_idx) {
        // Natural index (0, N) -> (block_idx, voxel_idx)
        int64_t workload_block_idx = workload_idx / resolution3;
        int64_t voxel_idx = workload_idx % resolution3;

        // voxel_idx -> (x_voxel, y_voxel, z_voxel)
        int64_t xv, yv, zv;
        voxel_indexer.WorkloadToCoord(voxel_idx, &xv, &yv, &zv);

        // Obtain voxel's mesh struct ptr
        int64_t workload_mesh_struct_idx;
        mesh_structure_indexer.CoordToWorkload(xv, yv, zv, workload_block_idx,
                                               &workload_mesh_struct_idx);
        int* mesh_struct_ptr =
                static_cast<int*>(mesh_structure_indexer.GetDataPtrFromWorkload(
                        workload_mesh_struct_idx));

        int table_idx = mesh_struct_ptr[3];
        if (tri_count[table_idx] == 0) return;

        for (size_t tri = 0; tri < 16; tri += 3) {
            if (tri_table[table_idx][tri] == -1) return;

            int tri_idx = atomicAdd(tri_count_ptr, 1);

            for (size_t vertex = 0; vertex < 3; ++vertex) {
                int edge = tri_table[table_idx][tri + vertex];

                int64_t xv_i = xv + edge_shifts[edge][0];
                int64_t yv_i = yv + edge_shifts[edge][1];
                int64_t zv_i = zv + edge_shifts[edge][2];
                int64_t edge_i = edge_shifts[edge][3];

                int dxb = xv_i / resolution;
                int dyb = yv_i / resolution;
                int dzb = zv_i / resolution;

                int nb_idx = (dxb + 1) + (dyb + 1) * 3 + (dzb + 1) * 9;

                int64_t block_idx_i =
                        nb_indices_ptr[nb_idx * n_blocks + workload_block_idx];
                int64_t workload_mesh_struct_i;
                mesh_structure_indexer.CoordToWorkload(
                        xv_i - dxb * resolution, yv_i - dyb * resolution,
                        zv_i - dzb * resolution, inv_indices_ptr[block_idx_i],
                        &workload_mesh_struct_i);
                if (indices_ptr[inv_indices_ptr[block_idx_i]] != block_idx_i) {
                    printf("inv indices error!\n");
                }
                int* mesh_struct_ptr_i = static_cast<int*>(
                        mesh_structure_indexer.GetDataPtrFromWorkload(
                                workload_mesh_struct_i));

                if (mesh_struct_ptr_i[edge_i] < 0) {
                    printf("triangle: mesh struct error");
                }
                triangles_ptr[3 * tri_idx + 2 - vertex] =
                        mesh_struct_ptr_i[edge_i];
            }
        }
    });

    int total_tri_count = triangle_count.Item<int>();
    utility::LogInfo("Total triangle count = {}", total_tri_count);
    triangles = triangles.Slice(0, 0, total_tri_count);
    dsts.emplace("triangles", triangles);
}

void GeneralEWCUDA(const std::unordered_map<std::string, Tensor>& srcs,
                   std::unordered_map<std::string, Tensor>& dsts,
                   GeneralEWOpCode op_code) {
    switch (op_code) {
        case GeneralEWOpCode::Unproject:
            CUDAUnprojectKernel(srcs, dsts);
            break;
        case GeneralEWOpCode::TSDFTouch:
            CUDATSDFTouchKernel(srcs, dsts);
            break;
        case GeneralEWOpCode::TSDFIntegrate:
            CUDATSDFIntegrateKernel(srcs, dsts);
            break;
        case GeneralEWOpCode::TSDFSurfaceExtraction:
            CUDASurfaceExtractionKernel(srcs, dsts);
            break;
        case GeneralEWOpCode::MarchingCubes:
            CUDAMarchingCubesKernel(srcs, dsts);
            break;
        case GeneralEWOpCode::RayCasting:
            break;
        case GeneralEWOpCode::Debug: {
            int64_t n = 10;
            CUDALauncher::LaunchGeneralKernel(
                    n, [=] OPEN3D_HOST_DEVICE(int64_t workload_idx) {});
            break;
        }
        default:
            break;
    }
}

}  // namespace kernel
}  // namespace core
}  // namespace open3d
