// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

// https://
// software.intel.com/sites/products/documentation/doclib/mkl_sa/11/mkl_lapack_examples/lapacke_sgesv_row.c.htm
#include <stdio.h>
#include <stdlib.h>

#include "open3d/core/op/linalg/LinalgUtils.h"
#include "open3d/core/op/linalg/Solve.h"

namespace open3d {
namespace core {

void SVDCUDA(const void* A_data,
             void* U_data,
             void* S_data,
             void* VT_data,
             void* superb_data,
             int m,
             int n,
             Dtype dtype,
             const Device& device) {
    hipsolverHandle_t handle = CuSolverContext::GetInstance()->GetHandle();
    int* dinfo = static_cast<int*>(MemoryManager::Malloc(sizeof(int), device));
    int len;

    switch (dtype) {
        case Dtype::Float32: {
            OPEN3D_CUSOLVER_CHECK(
                    hipsolverDnSgesvd_bufferSize(handle, m, n, &len),
                    "hipsolverDnSgesvd_bufferSize failed");
            void* workspace =
                    MemoryManager::Malloc(len * sizeof(float), device);

            OPEN3D_CUSOLVER_CHECK_WITH_DINFO(
                    hipsolverDnSgesvd(handle, 'A', 'A', m, n,
                                     const_cast<float*>(
                                             static_cast<const float*>(A_data)),
                                     m, static_cast<float*>(S_data),
                                     static_cast<float*>(U_data), m,
                                     static_cast<float*>(VT_data), n,
                                     static_cast<float*>(workspace), len,
                                     static_cast<float*>(superb_data), dinfo),
                    "hipsolverDnSgesvd failed with dinfo = ", dinfo, device);

            MemoryManager::Free(workspace, Device("CUDA:0"));
            break;
        }

        case Dtype::Float64: {
            OPEN3D_CUSOLVER_CHECK(
                    hipsolverDnDgesvd_bufferSize(handle, m, n, &len),
                    "hipsolverDnDgesvd_bufferSize failed");
            void* workspace =
                    MemoryManager::Malloc(len * sizeof(double), device);

            OPEN3D_CUSOLVER_CHECK_WITH_DINFO(
                    hipsolverDnDgesvd(
                            handle, 'A', 'A', m, n,
                            const_cast<double*>(
                                    static_cast<const double*>(A_data)),
                            m, static_cast<double*>(S_data),
                            static_cast<double*>(U_data), m,
                            static_cast<double*>(VT_data), n,
                            static_cast<double*>(workspace), len,
                            static_cast<double*>(superb_data), dinfo),
                    "hipsolverDnDgesvd failed with dinfo = ", dinfo, device);

            MemoryManager::Free(workspace, device);
            break;
        }

        default: {  // should never reach here
            utility::LogError("Unsupported dtype {} in SVDCUDA.",
                              DtypeUtil::ToString(dtype));
        }
    }

    MemoryManager::Free(dinfo, device);
}

}  // namespace core
}  // namespace open3d
