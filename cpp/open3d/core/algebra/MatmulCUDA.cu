// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include <hipblas.h>

#include "Matmul.h"
namespace open3d {
namespace core {
namespace _detail {

// Reference
// https://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemmbatched
// https://developer.nvidia.com/sites/default/files/akamai/cuda/files/Misc/mygpu.pdf

Tensor MatmulCUDA(const Tensor& A, const Tensor& B) {
    // Check dimensions
    SizeVector A_shape = A.GetShape();
    SizeVector B_shape = B.GetShape();

    if (A_shape.size() != 2) {
        utility::LogError("Tensor A must be 2D, but got {}D", A_shape.size());
    }
    if (B_shape.size() != 2) {
        utility::LogError("Tensor B must be 2D, but got {}D", B_shape.size());
    }
    if (A_shape[1] != B_shape[0]) {
        utility::LogError("Tensor A columns {} mismatch with Tensor B rows {}",
                          A_shape[1], B_shape[0]);
    }

    int64_t m = A_shape[0], k = A_shape[1], n = B_shape[1];

    // TODO: dtype and device check

    Tensor C = Tensor::Zeros({m, n}, A.GetDtype(), A.GetDevice());

    hipblasStatus_t stat;
    hipblasHandle_t handle;

    float alpha = 1.0f;
    float beta = 0.0f;

    void* A_data = A.Contiguous().GetDataPtr();
    void* B_data = B.Contiguous().GetDataPtr();
    void* C_data = C.GetDataPtr();

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        utility::LogError("Unable to create cublas handle");
    }

    // A: m x k
    // B: k x n
    // C: m x n
    // clang-format off
    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N, // A, B transpose flag
                m, n, k, // dimensions
                &alpha,
                static_cast<const float*>(A_data), m,
                static_cast<const float*>(B_data), k, // input and their leading dims
                &beta,
                static_cast<float*>(C_data), m); // output and its leading dim
    // clang-format on

    hipblasDestroy(handle);

    return C;
}
}  // namespace _detail
}  // namespace core
}  // namespace open3d
