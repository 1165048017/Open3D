#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2020 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include <iostream>

#include "open3d/ml/impl/misc/Nms.h"
#include "open3d/ml/pytorch/misc/NmsOpKernel.h"
#include "torch/script.h"

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x, " must be a CUDAtensor ")
#define CHECK_CONTIGUOUS(x) \
    TORCH_CHECK(x.is_contiguous(), #x, " must be contiguous ")
#define CHECK_INPUT(x) \
    CHECK_CUDA(x);     \
    CHECK_CONTIGUOUS(x)

const int THREADS_PER_BLOCK_NMS = sizeof(unsigned long long) * 8;

#define CHECK_ERROR(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code,
                      const char *file,
                      int line,
                      bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
                line);
        if (abort) exit(code);
    }
}

#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

int64_t NmsCUDA(torch::Tensor boxes,
                torch::Tensor keep,
                double nms_overlap_thresh) {
    // params boxes: (N, 5) [x1, y1, x2, y2, ry]
    // params keep: (N)
    printf("01!\n");

    CHECK_INPUT(boxes);
    CHECK_CONTIGUOUS(keep);

    int boxes_num = boxes.size(0);
    const float *boxes_data = boxes.data_ptr<float>();
    int64_t *keep_data = keep.data_ptr<int64_t>();

    const int col_blocks = DIVUP(boxes_num, THREADS_PER_BLOCK_NMS);

    printf("02!\n");

    unsigned long long *mask_data = NULL;
    CHECK_ERROR(
            hipMalloc((void **)&mask_data,
                       boxes_num * col_blocks * sizeof(unsigned long long)));
    open3d::ml::impl::NmsCUDAKernel(boxes_data, mask_data, boxes_num,
                                    nms_overlap_thresh);

    std::vector<unsigned long long> mask_cpu(boxes_num * col_blocks);

    CHECK_ERROR(hipMemcpy(&mask_cpu[0], mask_data,
                           boxes_num * col_blocks * sizeof(unsigned long long),
                           hipMemcpyDeviceToHost));
    printf("03!\n");

    hipFree(mask_data);

    unsigned long long remv_cpu[col_blocks];
    memset(remv_cpu, 0, col_blocks * sizeof(unsigned long long));

    int num_to_keep = 0;
    std::cout << "boxes_num: " << boxes_num << std::endl;

    for (int i = 0; i < boxes_num; i++) {
        std::cout << "i: " << i << std::endl;
        int nblock = i / THREADS_PER_BLOCK_NMS;
        int inblock = i % THREADS_PER_BLOCK_NMS;

        std::cout << "nblock: " << nblock << std::endl;
        std::cout << "inblock: " << inblock << std::endl;

        if (!(remv_cpu[nblock] & (1ULL << inblock))) {
            std::cout << "inside: 01" << std::endl;
            keep_data[num_to_keep++] = i;
            std::cout << "inside: 02" << std::endl;
            unsigned long long *p = &mask_cpu[0] + i * col_blocks;
            std::cout << "inside: 03" << std::endl;
            for (int j = nblock; j < col_blocks; j++) {
                remv_cpu[j] |= p[j];
            }
        }
    }
    printf("04!\n");

    if (hipSuccess != hipGetLastError()) {
        printf("Error!\n");
    }
    printf("05!\n");

    return num_to_keep;
}
